#include "hip/hip_runtime.h"
/*
 * src/matrix.cu: Matrix addition and subtraction
 *
 * Author: Tianyang Zhou <t7zhou@ucsd.edu>
 *
 */

#include <iostream>

#include "matrix.cuh"
#include "matrix.h"

#define THREADS_PER_BLOCK 1024

__global__ void GPU_add(double *A, double *B, double *C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        C[i] = A[i] + B[i];
    }

    return;
}

__global__ void GPU_sub(double *A, double *B, double *C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        C[i] = A[i] - B[i];
    }

    return;
}

__global__ void GPU_dot_product(double *A, double *B, double *C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        C[i] = A[i] * B[i];
    }

    return;
}

__global__ void GPU_transpose(double *A, double *B, int width, int height,
                              int base) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + base;

    int ix = i % width;
    int iy = i / width;

    if (ix < width && iy < height) {
        B[ix * height + iy] = A[i];
    }

    return;
}

__global__ void GPU_multiply(double *A, double *B, double *C, int n) {
    // use share
    return;
}

Matrix Matrix::gpu_add(const Matrix &other) {
    dim3 dimBlock(THREADS_PER_BLOCK);
    int size = this->width * this->height;
    dim3 dimGrid((size + dimBlock.x - 1) / dimBlock.x);

    Matrix result(this->height, this->width);

    // malloc device memory
    double *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size * sizeof(double));
    hipMalloc((void **)&d_B, size * sizeof(double));
    hipMalloc((void **)&d_C, size * sizeof(double));

    // copy data to device
    hipMemcpy(d_A, this->data, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, other.data, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, result.data, size * sizeof(double), hipMemcpyHostToDevice);

    GPU_add<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, size);

    // copy data back to host
    hipMemcpy(result.data, d_C, size * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);

    return result;
}

Matrix Matrix::gpu_sub(const Matrix &other) {
    dim3 dimBlock(THREADS_PER_BLOCK);
    int size = this->width * this->height;
    dim3 dimGrid((size + dimBlock.x - 1) / dimBlock.x);

    Matrix result(this->height, this->width);

    // malloc device memory
    double *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size * sizeof(double));
    hipMalloc((void **)&d_B, size * sizeof(double));
    hipMalloc((void **)&d_C, size * sizeof(double));

    // copy data to device
    hipMemcpy(d_A, this->data, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, other.data, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, result.data, size * sizeof(double), hipMemcpyHostToDevice);

    GPU_sub<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, size);

    // copy data back to host
    hipMemcpy(result.data, d_C, size * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);

    return result;
}

Matrix Matrix::gpu_dot_product(const Matrix &other) {
    dim3 dimBlock(THREADS_PER_BLOCK);
    int size = this->width * this->height;
    dim3 dimGrid((size + dimBlock.x - 1) / dimBlock.x);

    Matrix result(this->height, this->width);

    // malloc device memory
    double *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size * sizeof(double));
    hipMalloc((void **)&d_B, size * sizeof(double));
    hipMalloc((void **)&d_C, size * sizeof(double));

    // copy data to device
    hipMemcpy(d_A, this->data, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, other.data, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, result.data, size * sizeof(double), hipMemcpyHostToDevice);

    GPU_dot_product<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, size);

    // copy data back to host
    hipMemcpy(result.data, d_C, size * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);

    return result;
}

Matrix Matrix::gpu_transpose() {
    dim3 dimBlock(THREADS_PER_BLOCK);
    int size = this->width * this->height;
    dim3 dimGrid((size + dimBlock.x - 1) / dimBlock.x);
    Matrix result(this->width, this->height);

    // malloc device memory
    double *d_A, *d_B;
    hipMalloc((void **)&d_A, size * sizeof(double));
    hipMalloc((void **)&d_B, size * sizeof(double));

    // implement transpose by 4 streams
    hipStream_t stream[4];
    for (int i = 0; i < 4; i++) {
        hipStreamCreate(&stream[i]);
    }
    for (int i = 0; i < 4; i++) {
        hipMemcpyAsync(d_A + i * size / 4, this->data + i * size / 4,
                        size * sizeof(double), hipMemcpyHostToDevice,
                        stream[i]);

        GPU_transpose<<<dimGrid, dimBlock.x / 4, 0, stream[i]>>>(
            d_A, d_B, this->width, this->height, i * size / 4);

        hipMemcpyAsync(result.data + i * size / 4, d_B + i * size / 4,
                        size * sizeof(double) / 4, hipMemcpyDeviceToHost,
                        stream[i]);
    }
    // sync and free streams
    for (int i = 0; i < 4; i++) {
        hipStreamSynchronize(stream[i]);
        hipStreamDestroy(stream[i]);
    }

    // free device memory
    hipFree(d_A);
    hipFree(d_B);

    return result;
}

Matrix Matrix::gpu_mul(const Matrix &other) {
    dim3 dimBlock(THREADS_PER_BLOCK);
    int size = this->width * this->height;
    dim3 dimGrid((size + dimBlock.x - 1) / dimBlock.x);

    Matrix result(this->height, this->width);

    // malloc device memory
    double *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size * sizeof(double));
    hipMalloc((void **)&d_B, size * sizeof(double));
    hipMalloc((void **)&d_C, size * sizeof(double));

    // copy data to device
    hipMemcpy(d_A, this->data, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, other.data, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, result.data, size * sizeof(double), hipMemcpyHostToDevice);

    GPU_multiply<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, size);
    // TODO:
}
